#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "timestamp.h"

#define CONVERGE_CHECK_TRUE

#define CUDA_SAFE_CALL(call)                                                  \
  {                                                                           \
    hipError_t err = call;                                                     \
    if (hipSuccess != err)                                                   \
    {                                                                         \
      fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", __FILE__, \
              __LINE__, hipGetErrorString(err));                             \
      exit(EXIT_FAILURE);                                                     \
    }                                                                         \
  }

#define FRACTION_CEILING(numerator, denominator) \
  ((numerator + denominator - 1) / denominator)

// declare constant-device variables
__constant__ int n, m, maxXCount, maxYCount;
__constant__ double relax, cx_cc, cy_cc, c1, c2, xLeft, xRight, yBottom, yUp, deltaX, deltaY;

int h_n, h_m, h_maxXCount, h_maxYCount;
double h_relax, h_cx_cc, h_cy_cc, h_c1, h_c2, h_xLeft, h_xRight, h_yBottom, h_yUp, h_deltaX, h_deltaY;

// ON-HOST FUNCTIONS

// solution checker
double checkSolution(double xStart, double yStart, int maxXCount, int maxYCount,
                     double *u, double deltaX, double deltaY, double alpha);

void initGPU(void);

// ON-DEVICE FUNCTIONS

__global__ void kernel(double *u, double *u_old, double *error_matrix)
{
  // calculate x and y before do the following line
  int ti = threadIdx.x + blockIdx.x * blockDim.x; // get thread id

  if (ti >= n * m) // Required in cases where the number of elements
    return;        // is *not* a multiple of threads per block (aka 1024) eg. 1680x1680/1024=2756.25 -> 2757 blocks

  int x = (ti % m);
  int y = (ti / n);

  /////////////////////////////

  // u_temp : [0, (Bdim + 2) * 3 - 1];
  extern __shared__ double u_tmp[];

  // we spawn n*m threads,
  // map "index" from indexing n*m elements -> (n+2)*(m+2) elements, including halos
  int index = ti + (m + 2) + 2 * (ti / m + 1) - 1;

  if (threadIdx.x == 0)
  {                                           // 1st element
    u_tmp[blockDim.x + 2] = u_old[index - 1]; // center left
  }

  if (threadIdx.x == blockDim.x - 1)
  {                                               // last element
    u_tmp[2 * blockDim.x + 3] = u_old[index + 1]; // center right
  }

  u_tmp[1 + threadIdx.x] = u_old[index - (m + 2)];    // upper
  u_tmp[blockDim.x + 3 + threadIdx.x] = u_old[index]; // center

  // if (index + m + 2 >= (n+2)*(m+2)) printf("$$$someone fucked up n = %d, m = %d -- %d %d %d %d %d %d %d\n",
  // n, m, (n*m), ((n+2)*(m+2)), index, threadIdx.x, blockIdx.x, blockDim.x, gridDim.x);
  // if (2*blockDim.x + 5 + threadIdx.x >= ((1024 + 2) * 3)) printf("$$$$wat\n");

  u_tmp[2 * blockDim.x + 5 + threadIdx.x] = u_old[index + (m + 2)]; // lower

  // u_tmp[3][Bdim+2]
  // [0, Bdim+1] // upper row
  // [Bdim+2, Bdim+2 + (Bdim+1)] // center row
  // [2Bdim+4, 2Bim+4 + (Bdim+1)] // lower row
  // Tid in [0, Bdim-1]

  double fX = (xLeft + (x - 1) * deltaX), fY = (yBottom + (y - 1) * deltaY);
  double fX_sq = fX * fX, fY_sq = fY * fY;
  double fX_dot_fY_sq = fX_sq * fY_sq;
  int tmp_index = ((blockDim.x + 2) * 3);

  __syncthreads();

  // do calculations

  double updateVal = (u_tmp[blockDim.x + threadIdx.x + 2] + u_tmp[blockDim.x + threadIdx.x + 4]) * cx_cc + // left, right
                     (u_tmp[1 + threadIdx.x] + u_tmp[2 * blockDim.x + threadIdx.x + 5]) * cy_cc +          // up, down
                     u_tmp[blockDim.x + threadIdx.x + 3] +                                                 // self
                     c1 * (1.0 - fX_sq - fY_sq + fX_dot_fY_sq) -
                     c2 * (fX_dot_fY_sq - 1.0);

  u_tmp[tmp_index + threadIdx.x] = updateVal * updateVal;
  u[index] = u_tmp[blockDim.x + threadIdx.x + 3] - relax * updateVal;
  
  int stride = blockDim.x / 2;

  __syncthreads();

  while (stride > 0)
  {   
    if (threadIdx.x > stride) // Required in cases where the number of elements
      break;         // is *not* a multiple of threads per block (aka 1024) eg. 1680x1680/1024=2756.25 -> 2757 blocks

    u_tmp[tmp_index + threadIdx.x] += u_tmp[tmp_index + threadIdx.x + stride - 1];
    stride >>= 1;
  }

  if (threadIdx.x == 0) {
    error_matrix[blockIdx.x] = u_tmp[tmp_index];
  }
}

// NOTE: na valoume kai ton ari8miti apo to stride
__global__ void kernel_reduce_error(double *error_matrix, int stride)
{
  int ti = threadIdx.x + blockIdx.x * blockDim.x; // get thread id

  if (ti >= stride) // Required in cases where the number of elements
    return;         // is *not* a multiple of threads per block (aka 1024) eg. 1680x1680/1024=2756.25 -> 2757 blocks

  error_matrix[ti] = error_matrix[ti] + error_matrix[ti + stride];
}

int main(int argc, char **argv)
{
  int mits, allocCount, iterationCount, maxIterationCount, stride;
  double alpha, tol, maxAcceptableError, error;
  double *u, *u_old, *tmp, *error_matrix;
  // double t1, t2;

  //    printf("Input n,m - grid dimension in x,y direction:\n");
  scanf("%d,%d", &h_n, &h_m);
  //    printf("Input alpha - Helmholtz constant:\n");
  scanf("%lf", &alpha);
  //    printf("Input relax - successive over-relaxation parameter:\n");
  scanf("%lf", &h_relax);
  //    printf("Input tol - error tolerance for the iterrative solver:\n");
  scanf("%lf", &tol);
  //    printf("Input mits - maximum solver iterations:\n");
  scanf("%d", &mits);

  printf("-> %d, %d, %g, %g, %g, %d\n", h_n, h_m, alpha, h_relax, tol, mits);

  allocCount = (h_n + 2) * (h_m + 2);

 
  ////////////////////////////////
  // Cuda malloc test
  double *h_u, *h_u_old, *h_error_matrix;
  h_u = (double *) calloc(allocCount, sizeof(double));
  h_u_old = (double *) calloc(allocCount, sizeof(double));
  h_error_matrix = (double *) calloc(allocCount, sizeof(double));
  
  CUDA_SAFE_CALL(hipMalloc(&u, allocCount * sizeof(double)));
  CUDA_SAFE_CALL(hipMalloc(&u_old, allocCount * sizeof(double)));
  CUDA_SAFE_CALL(hipMalloc(&error_matrix, allocCount * sizeof(double))); 
  CUDA_SAFE_CALL(hipMemcpy(u, h_u, allocCount * sizeof(double), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(u_old, h_u_old, allocCount * sizeof(double), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(error_matrix, h_error_matrix, allocCount * sizeof(double), hipMemcpyHostToDevice));

  maxIterationCount = mits;
  maxAcceptableError = tol;

  // Solve in [-1, 1] x [-1, 1]
  h_xLeft = h_yBottom = -1.0;
  h_xRight = h_yUp = 1.0;

  h_deltaX = (h_xRight - h_xLeft) / (h_n - 1);
  h_deltaY = (h_yUp - h_yBottom) / (h_m - 1);

  iterationCount = 0;
  error = HUGE_VAL;

  // clock_t start = clock(), diff;
  //   t1 = MPI_Wtime();

  timestamp t_start;
  t_start = getTimestamp();

  h_maxXCount = h_n + 2;
  h_maxYCount = h_m + 2;

  double cx = 1.0 / (h_deltaX * h_deltaX);
  double cy = 1.0 / (h_deltaY * h_deltaY);
  double cc = -2.0 * (cx + cy) - alpha;
  double div_cc = 1.0 / cc;

  h_cx_cc = 1.0 / (h_deltaX * h_deltaX) * div_cc;
  h_cy_cc = 1.0 / (h_deltaY * h_deltaY) * div_cc;
  h_c1 = (2.0 + alpha) * div_cc;
  h_c2 = 2.0 * div_cc;

  // pass_values_to_gpu();
  initGPU();

  // set blocks and threads/block TODO: make it more generic
  int BLOCK_SIZE = 128;
  printf("GPU Threads used per block: %d\n", BLOCK_SIZE);
  dim3 dimBl(BLOCK_SIZE);
  dim3 dimGr(FRACTION_CEILING(h_n * h_m, BLOCK_SIZE));

  /* Iterate as long as it takes to meet the convergence criterion */
  while (iterationCount < maxIterationCount && error > maxAcceptableError)
  {
    iterationCount++;

    /*************************************************************
     * Performs one iteration of the Jacobi method and computes
     * the residual value.
     *
     * NOTE: u(0,*), u(maxXCount-1,*), u(*,0) and u(*,maxYCount-1)
     * are BOUNDARIES and therefore not part of the solution.
     *************************************************************/

    // error = 0.0;

    // run kernel
    kernel<<<dimGr, dimBl, ((BLOCK_SIZE + 2) * 3 + BLOCK_SIZE + 1) * sizeof(double)>>>(u, u_old, error_matrix); //xd /bruh

    // estimate the error : error += updateVal * updateVal;

    CUDA_SAFE_CALL(hipDeviceSynchronize());

    // error = sqrt(error) / (n * m);

  #ifdef CONVERGE_CHECK_TRUE
    stride = h_n * h_m / 2;
    while (stride > 0)
    {
      int BLOCK_SIZE = ( (stride < 128 ? stride : 128) );
      dim3 dimBl(BLOCK_SIZE);
      dim3 dimGr(FRACTION_CEILING(stride, BLOCK_SIZE));

      kernel_reduce_error<<<dimGr, dimBl>>>(error_matrix, stride);
      CUDA_SAFE_CALL(hipDeviceSynchronize());
      stride >>= 1;
    }

    CUDA_SAFE_CALL(hipMemcpy(h_error_matrix, error_matrix, sizeof(double), hipMemcpyDeviceToHost));

    error = sqrt(h_error_matrix[0]) / (h_n * h_m);
  #endif
  
    // Swap the buffers
    tmp = u_old;
    u_old = u;
    u = tmp;
  }

  float msec = getElapsedtime(t_start);

  printf("Time taken: %f seconds\n", msec / 1000.0);
  printf("Iterations: %d\n", iterationCount);
  printf("Residual: %g\n", error); // :(

  CUDA_SAFE_CALL(hipMemcpy(h_u_old, u_old, allocCount * sizeof(double), hipMemcpyDeviceToHost));

  // u_old holds the solution after the most recent buffers swap
  double absoluteError =
      checkSolution(h_xLeft, h_yBottom, h_n + 2, h_m + 2, h_u_old, h_deltaX, h_deltaY, alpha);
  printf("The error of the iterative solution is %g\n", absoluteError);

  free(h_u);
  free(h_u_old);
  free(h_error_matrix);

  CUDA_SAFE_CALL(hipFree(u));
  CUDA_SAFE_CALL(hipFree(u_old));
  CUDA_SAFE_CALL(hipFree(error_matrix));
  return 0;
}

/**********************************************************
 * Checks the error between numerical and exact solutions
 **********************************************************/
double checkSolution(double xStart, double yStart, int maxXCount, int maxYCount,
                     double *u, double deltaX, double deltaY, double alpha)
{
#define U(XX, YY) u[(YY)*maxXCount + (XX)]
  int x, y;
  double fX, fY;
  double localError, error = 0.0;

  for (y = 1; y < (maxYCount - 1); y++)
  {
    fY = yStart + (y - 1) * deltaY;
    for (x = 1; x < (maxXCount - 1); x++)
    {
      fX = xStart + (x - 1) * deltaX;
      localError = U(x, y) - (1.0 - fX * fX) * (1.0 - fY * fY);
      error += localError * localError;
    }
  }
  return sqrt(error) / ((maxXCount - 2) * (maxYCount - 2));
}

void initGPU(void)
{ // bruh
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(n), &h_n, sizeof(int), 0, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(m), &h_m, sizeof(int), 0, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(maxXCount), &h_maxXCount, sizeof(int), 0, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(maxYCount), &h_maxYCount, sizeof(int), 0, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(relax), &h_relax, sizeof(double), 0, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cx_cc), &h_cx_cc, sizeof(double), 0, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cy_cc), &h_cy_cc, sizeof(double), 0, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c1), &h_c1, sizeof(double), 0, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c2), &h_c2, sizeof(double), 0, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(xLeft), &h_xLeft, sizeof(double), 0, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(xRight), &h_xRight, sizeof(double), 0, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(yBottom), &h_yBottom, sizeof(double), 0, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(yUp), &h_yUp, sizeof(double), 0, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(deltaX), &h_deltaX, sizeof(double), 0, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(deltaY), &h_deltaY, sizeof(double), 0, hipMemcpyHostToDevice));
}
